#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gaincorr.h"
#include "math.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <list>
#include"book.cuh"

__global__ void kernel_do_gaincorr (float* d_slope, float* d_intercept, int* d_saturation, float* d_image)
{

    unsigned int tid = threadIdx.x;
    unsigned int pixel = blockIdx.x*blockDim.x + tid; //thread is computing pixel-th pixel

    //printf(" pixel: %d \t tid: %d \t blockIdx : %d \t blockDim : %d \n", pixel,tid, blockIdx.x, blockDim.x);



        d_image[pixel] = (d_image[pixel] - d_intercept[pixel] ) / d_slope[pixel]  * 16383.0f / *d_saturation;
    return;
}







void Gaincorr::gaincorrigateimage(Image_cuda_compatible& image)
{
    //Rounding voltage to multiply of 5
    int voltage = (int) (round(image.getvoltage()));
    int remainder  = voltage %5;

    if(remainder != 0)

    {
        voltage = voltage + 5 - remainder;
    }

//DEBUG
    if(saturation.find(voltage) == saturation.end())
    {
        std::cout <<"Error: no calbration data found for image" <<image.getid() << "With voltage " << image.getvoltage() <<std::endl;
        return;
    }

    if(slopes.find(voltage) == slopes.end())
    {
        std::cout <<"Error: no slope data found for image" <<image.getid() << "With voltage " << image.getvoltage() <<std::endl;
        return;
    }

    if(intercepts.find(voltage) == intercepts.end())
    {
        std::cout <<"Error: no slope data found for image" <<image.getid() << "With voltage " << image.getvoltage() <<std::endl;
        return;
    }



    int* d_saturation;
    //DEBUG
    int sat = saturation[voltage];
    float* d_slope;
    float* d_intercept;
    d_slope = slopes.find(voltage)->second.gpu_im;
    d_intercept = intercepts.find(voltage)->second.gpu_im;
    float* d_image;

    d_image= image.gpu_im;

   /*
    std::cout << "GAIN CORRECTION" <<std::endl<<std::endl;
   std::cout << "Image:" <<std::endl;
    std::cout << "VOltage: " << image.getvoltage();
   std::cout << "Min: " << image.getmin() <<"\t Mean: " << image.getmean()
             <<"\t Max: " << image.getmax() <<std::endl <<std::endl;
   std::cout << "slope:" <<std::endl;
   std::cout << "Min: " << slopes.find(voltage)->second.getmin() <<"\t Mean: " << slopes.find(voltage)->second.getmean()
             <<"\t Max: " << slopes.find(voltage)->second.getmax() <<std::endl <<std::endl;
   std::cout << "Intercept:" <<std::endl;
   std::cout << "Min: " << intercepts.find(voltage)->second.getmin() <<"\t Mean: " << intercepts.find(voltage)->second.getmean()
             <<"\t Max: " << intercepts.find(voltage)->second.getmax() <<std::endl <<std::endl;
   std::cout <<"Saturation: " << sat << std::endl << std::endl;
   */





    HANDLE_ERROR (hipMalloc( (void**)&d_saturation, sizeof(int) ));
   HANDLE_ERROR (hipMemcpy(d_saturation, &sat, sizeof(int), hipMemcpyHostToDevice ));

    kernel_do_gaincorr<<<41472,32>>>( d_slope,  d_intercept, d_saturation,  d_image );



    HANDLE_ERROR (hipFree(d_saturation));

}






