#include "hip/hip_runtime.h"
#include ""
#include "gc_im_container.h"

gc_im_container::~gc_im_container()
{
   removefromgpu();
   removefromhost();
}

void  gc_im_container::inicialize(int n)
{
    removefromhost();

    if( n > 0)
    {
        size = n;
        hipHostAlloc( (void**) &h_images, sizeof(float) *1327104 * size, hipHostMallocDefault);
        hipHostAlloc( (void**) &h_settings, sizeof(float)  * size, hipHostMallocDefault);

        hipMemset(&h_images, 0,sizeof(float) *1327104 * size );

        hipMemset(&h_settings, 0,sizeof(float)  * size );
    }
return;
}

void gc_im_container::removefromgpu()
{
    if(d_images != NULL)
    {
        hipFree(d_images);
        d_images = NULL;
    }
    if(d_settings != NULL)
    {
        hipFree(d_settings);
        d_settings = NULL;
    }
}

void gc_im_container::removefromhost()
{
    if(h_images != NULL)

    {
        hipHostFree(h_images);
        h_images = NULL;
    }
    if(h_settings != NULL)
    {
        hipHostFree(h_settings);
        h_settings=NULL;
    }
}





void gc_im_container::copy_to_GPU(float*& d_im, float*& d_set)
{
    if(images > 0 && h_images != NULL && h_settings!= NULL)
    {
        removefromgpu();

        if(images > 0)
        {
            hipMalloc( (void**) &d_im, sizeof(float) *1327104 * size );
            hipMalloc( (void**) &d_set, sizeof(float)  * size );

            d_images = d_im;
            d_settings = d_settings;
        }
        hipMemcpy(d_images, h_images, sizeof(float) *1327104 * size , hipMemcpyHostToDevice );
        hipMemcpy(d_settings, h_settings, sizeof(float) * size , hipMemcpyHostToDevice );
    }

}

void gc_im_container::reserveIfEmpty(int n)
{
    if( size ==0)
    {

        inicialize(n);
    }
    return;
}


