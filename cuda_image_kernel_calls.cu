#include "cuda_reduce_kernels.cuh"
//#include "cuda_image_kernel_calls.h"
#include "image_cuda_compatible.h"
#include "book.h"




//! Copies image to the GPU and calculates the mean intensity on the GPU.
//! The image is than deleted from the GPU.
void Image_cuda_compatible::kernel_call_calculate_image_mean()
{
    long imagesize = size;
    unsigned short* d_image;
    hipMalloc( (void**)&d_image,imagesize*sizeof(unsigned short));
  double* d_data;
  hipMalloc( (void**)&d_data, sizeof(double) * 1024);
  double* d_sum;
  HANDLE_ERROR (hipMalloc( (void**)&d_sum, sizeof(double)));

  hipMemcpy(d_image,im,size * sizeof(unsigned short),hipMemcpyHostToDevice);
  kernel_reduce_sum_first_step<1024><<<64, 1024,  1024*sizeof(double)>>>(d_image,d_data, imagesize);
  kernel_reduce_sum_second_step<64><<<1,64, 64*sizeof(double)>>>(d_data, d_sum);
  double *h_sum;
h_sum = (double*) malloc(sizeof(double));
HANDLE_ERROR (hipMemcpy(h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost));
 mean = (float) ( (*h_sum )/ imagesize);



free(h_sum);
  hipFree(d_image);
  hipFree(d_data);
  hipFree(d_sum);

}







