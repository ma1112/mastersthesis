#include "hip/hip_runtime.h"
#include "cuda_reduce_kernels.cuh"
//#include "cuda_image_kernel_calls.h"
#include "image_cuda_compatible.h"
#include "book.h"
#include <stdio.h>
#include <iostream>
#include <string>


//! Kernel to add another image's pixel values to this image.
__global__ void kernel_addImage(float* d_this, float* d_other)
{

    unsigned int pixel = blockIdx.x*blockDim.x + threadIdx.x; //thread is computing pixel-th pixel
    d_this[pixel] += d_other[pixel];
    return;

}

//! Kernel to substract another image's pixel values from this image.

__global__ void kernel_subtractImage(float* d_this, float* d_other)
{

    unsigned int pixel = blockIdx.x*blockDim.x + threadIdx.x; //thread is computing pixel-th pixel
    d_this[pixel] -= d_other[pixel];
    return;

}



//! Kernel to divide all pixel values by a float.
__global__ void kernel_divideImage(float* d_this, float divisor)
{

    unsigned int pixel = blockIdx.x*blockDim.x + threadIdx.x; //thread is computing pixel-th pixel
    d_this[pixel] /= divisor;
    return;
}



__global__ void kernel_multiplyImage(float* d_this, float multiplier)
{

    unsigned int pixel = blockIdx.x*blockDim.x + threadIdx.x; //thread is computing pixel-th pixel
    d_this[pixel] *= multiplier;
    return;
}

__global__ void kernel_loadFromUShortArray(unsigned short* d_ushort, float* d_image)
{
    unsigned int pixel = blockIdx.x*blockDim.x + threadIdx.x; //thread is computing pixel-th pixel
    d_image[pixel] = (float) d_ushort[pixel];
    return;
}


__global__ void kernel_exportToUSarray( float* d_image, unsigned short* d_ushort)
{
    unsigned int pixel = blockIdx.x*blockDim.x + threadIdx.x; //thread is computing pixel-th pixel
    d_ushort[pixel]   = (unsigned short) d_image[pixel];
    return;
}



//! Deassings memory from the GPU.
void Image_cuda_compatible::remove_from_GPU()
{
    if(gpu_im != NULL)
        {
       HANDLE_ERROR ( hipFree(gpu_im));
        gpu_im = NULL;
    }
}






//! Copies image to the GPU and calculates the mean intensity on the GPU.
void Image_cuda_compatible::calculate_meanvalue_on_GPU()
{
    if(gpu_im == NULL)
    {
        mean = 0.0f;
        return;
    }


  float* d_data;
  HANDLE_ERROR (hipMalloc( (void**)&d_data, 3*sizeof(float) * 1024));
  float *d_sum, *d_min, *d_max;
  HANDLE_ERROR (hipMalloc( (void**)&d_sum, sizeof(float)));
  HANDLE_ERROR (hipMalloc( (void**)&d_min, sizeof(float)));
  HANDLE_ERROR (hipMalloc( (void**)&d_max, sizeof(float)));


  kernel_reduce_sum_first_step<1024><<<64, 1024,  3*1024*sizeof(float)>>>(gpu_im, d_data, size);
  kernel_reduce_sum_second_step<64><<<1,64, 3*64*sizeof(float)>>>(d_data, d_sum, d_min, d_max);
  float *h_sum;
h_sum = (float*) malloc(sizeof(float));
HANDLE_ERROR (hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost));
 mean =  ( (*h_sum )/ size);
 HANDLE_ERROR (hipMemcpy(&min, d_min, sizeof(float), hipMemcpyDeviceToHost));
 HANDLE_ERROR (hipMemcpy(&max, d_max, sizeof(float), hipMemcpyDeviceToHost));




free(h_sum);
  HANDLE_ERROR(hipFree(d_data));
  HANDLE_ERROR(hipFree(d_sum));
  HANDLE_ERROR(hipFree(d_min));
  HANDLE_ERROR(hipFree(d_max));


}






//! Reserves memory for the image on the GPU.
float* Image_cuda_compatible::reserve_on_GPU()
{
    if( gpu_im == NULL)
    {
      //  std::cout << "gpu_im ==" << gpu_im <<" And now mallocing memory. "
       //           <<std::endl << "filename: " << filename << std::endl;

        HANDLE_ERROR( hipMalloc( (void**)&gpu_im,size*sizeof(float)));
     //   std::cout << "Malloc succesful & " << gpu_im <<std::endl;
       // std::cout << "Reserving memory on GPU for image "
                  //<<id << "at address @" << gpu_im <<std::endl;
       HANDLE_ERROR( hipMemset(gpu_im,0,size*sizeof(float)));
    }

    return gpu_im;
}


//! Copies an image from the GPU memory to this image's memory on the GPU.
float* Image_cuda_compatible::copy_GPU_image(float* other)
{
    reserve_on_GPU();
    if(other !=NULL)
    {
        //std::cout << "Copy image data from @" << other<< " to @" <<gpu_im <<std::endl;
        HANDLE_ERROR (hipMemcpy( gpu_im,other,size * sizeof(float),hipMemcpyDeviceToDevice));
    }
    else
    {
        std::cout <<"WARNING: The image you want to copy is not on the GPU." << std::endl;
    }
    return gpu_im;
}

//! Adds an image's pixel values to this image on the GPU.
void Image_cuda_compatible::add_on_GPU(Image_cuda_compatible &other)
{
//    std::cout << "Add_on_GPU()" << std::endl;

   //std::cout << "kernel_addimage (@" << gpu_im<< ", @" << other.gpu_im<<std::endl;
    kernel_addImage<<<2592,512>>>(gpu_im, other.gpu_im);
   // std::cout <<"done" << std::endl;
}

void Image_cuda_compatible::subtract_on_GPU(Image_cuda_compatible &other)
{

    kernel_subtractImage<<<2592,512>>>(gpu_im, other.gpu_im);

}

void Image_cuda_compatible::divide_on_GPU(float divisor)
{

    kernel_divideImage<<<2592,512>>>(gpu_im, divisor);

}

void Image_cuda_compatible::multiply_on_GPU(float multiplier)
{
    kernel_multiplyImage<<<2592,512>>>(gpu_im, multiplier);

}

void Image_cuda_compatible::cudaReadFromFile(const char* filename)
{

    FILE *file;
    file = fopen(filename,"rb");
    if (!file)
        {
                printf("Unable to open file! %s", filename);
                return;
        }
    unsigned short *temp, *d_temp;
    HANDLE_ERROR(hipHostAlloc((void**)&temp, size*sizeof(unsigned short), hipHostMallocDefault));
    fread(temp,sizeof(unsigned short),size,file);
    HANDLE_ERROR(hipMalloc((void**)&d_temp,size*sizeof(unsigned short)));
    HANDLE_ERROR(hipMemcpy(d_temp,temp,sizeof(unsigned short) * size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipHostFree(temp));
    kernel_loadFromUShortArray<<<2592,512>>>(d_temp, reserve_on_GPU());
    HANDLE_ERROR(hipFree(d_temp));
    fclose(file);


}


void Image_cuda_compatible::cudaReadFromFloatFile(const char* filename)
{
    FILE *file;
    file = fopen(filename,"rb");
    if (!file)
        {
                printf("Unable to open file! %s", filename);
                return;
        }
    float *temp;
    HANDLE_ERROR(hipHostAlloc((void**)&temp, size*sizeof(float), hipHostMallocDefault));
    fread(temp,sizeof(float),size,file);
    HANDLE_ERROR(hipMemcpy(reserve_on_GPU(),temp,sizeof(float) * size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipHostFree(temp));
    fclose(file);


}

void Image_cuda_compatible::cudaGetShortArrayToHost(unsigned short *h_sImage)
{
    unsigned short *d_usimage;
    HANDLE_ERROR(hipMalloc((void**) & d_usimage, size*sizeof(unsigned short)));
   kernel_exportToUSarray<<<2592,512>>>( gpu_im, d_usimage);
   HANDLE_ERROR(hipMemcpy(h_sImage, d_usimage, sizeof(unsigned short) * size , hipMemcpyDeviceToHost));
   HANDLE_ERROR(hipFree(d_usimage));
   return;
}

void Image_cuda_compatible::cudaGetArrayToHost(float *h_image)
{
    HANDLE_ERROR(hipMemcpy(h_image, gpu_im,sizeof(float) * size, hipMemcpyDeviceToHost));
    return;
}



//! Writes image values to a binary file, with unsigned int values.

void Image_cuda_compatible::writetofile(std::string filename)
{
    unsigned short* sh_im = new unsigned short[size];
    cudaGetShortArrayToHost(sh_im);


    FILE *file;


    file = fopen(filename.c_str(), "wb");
    if(file == NULL)
    {
            std::cout << "Failed to open file " << filename << "for writing."<< std::endl;
            return;
    }
    fwrite(sh_im, sizeof(unsigned short), size, file );
    delete[] sh_im;
    fclose(file);

}
